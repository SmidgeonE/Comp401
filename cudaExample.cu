#include "hip/hip_runtime.h"
#include <iostream>
#include <cudaExample.cuh>

__global__ void addThreeNumbers(int a, int b, int c, int *result) {
    *result = a + b + c;
}

int main() {
    int a = 5, b = 10, c = 15;
    int result;
    int *d_result;

    hipMalloc((void**)&d_result, sizeof(int));

    addThreeNumbers<<<1, 1>>>(a, b, c, d_result);

    hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "The result of adding " << a << ", " << b << ", and " << c << " is " << result << std::endl;

    hipFree(d_result);

    return 0;
}